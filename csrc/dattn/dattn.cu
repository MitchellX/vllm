#include "hip/hip_runtime.h"
/*
 Copyright (c) ByteDance Inc.
 Authors: 
  - Tongping Liu (tongping.liu@bytedance.com)
 */ 
 
#include <c10/core/ScalarType.h>
#include <cstdint>
#include <cstdio>
#include <string>
#include <hip/hip_runtime.h>
#include <Python.h>
#include <pthread.h>

#include "dattn.h"


#define KV_UTILIZATION_RATE (0.9)

static hipMemAllocationProp _prop = {};
static hipMemAccessDesc _accessDescr = {};
 
/* 
  In this allocator, we only have the following concepts, but without the concept of tokens.
  The python portion should convert the number of tokens to tokens depending on their block_size (e.g., 16)
  Region: virtual address space for a request. Currently, we support the space for max_seq_len.
 */
static uint64_t roundup(uint64_t size, uint64_t align_size) {
  return ((size + align_size - 1)/align_size) * align_size; 
}

static int allocatePhyPages(void * ptr, uint64_t size) {
  hipDeviceptr_t dptr = (hipDeviceptr_t)ptr;

  hipDevice_t dev; // device
  CHECK_DRV(hipCtxGetDevice(&dev));
  _prop.location.id = dev;
  _accessDescr.location = _prop.location;

  hipError_t status = hipSuccess;
  hipMemGenericAllocationHandle_t allocationHandle;
  if ((status = hipMemCreate(&allocationHandle, size, &_prop, 0)) == hipSuccess) {
    if ((status = hipMemMap(dptr, size, 0ULL, allocationHandle, 0ULL)) == hipSuccess) {
      if ((status = hipMemSetAccess(dptr, size, &_accessDescr, 1)) != hipSuccess) {
        fprintf(stderr, "hipMemMap success,but hipMemSetAccess failed!, err code: %d\n", status);
        hipMemUnmap(dptr, size);
      }
    }
    // always release the handle, but the memory is accessible util hipMemUnmap
    if((status = hipMemRelease(allocationHandle)) != hipSuccess) {
      fprintf(stderr, "hipMemRelease failed, err code: %d\n", status);
    } 
  } else {
    fprintf(stderr, "hipMemCreate %lx failed!, err code: %d\n", size, status);
  }
  return status == hipSuccess ? 0 : -1;
}

// Free the physical memory [ptr, ptr + size]
static void freePhysicalMemory(void* ptr, size_t size) {
  hipDeviceptr_t dptr = (hipDeviceptr_t)ptr;
  hipError_t res = hipMemUnmap(dptr, size); 
  if(res != hipSuccess) {
    const char* errorStr;
    hipDrvGetErrorString(res, &errorStr);
    fprintf(stderr, "hipMemUnmap failed when deallocating ptr %p and size %lx with error %s\n", ptr, size, errorStr);
  } 
}

/*
** kvCacheRegion functions implementation
*/
kvCacheRegion::kvCacheRegion(uint64_t region_size, uint64_t block_size, uint64_t page_size, hipDeviceptr_t ptr) {
  this->region_size = region_size;
  this->block_size = block_size;
  this->page_size = page_size; 
  this->dptr = reinterpret_cast<char*>(ptr);  
  this->nextUnmapedAddr = reinterpret_cast<char*>(ptr); 

  this->offset = 0; 
  this->total_pages = 0;
  this->used_pages = 0; 
}

// Decontructor: release all physical pages of this region
kvCacheRegion::~kvCacheRegion() {
  freeAllPhyMemory(); 
  // Note that since the region is detroyed, 
  // no need to clear other counters. 
}

void * kvCacheRegion::getStartPtr(void) {
  return reinterpret_cast<void*>(this->dptr); 
} 

uint64_t kvCacheRegion::getAllocPhyPages(void) {
  return this->total_pages;
} 

uint64_t kvCacheRegion::getUsedPhysicalPages(void) {
  return this->used_pages; 
}

/*
  kvCacheRegion function: allocate cached blocks  
    if the return value > 0, then it is succesful. 
 */ 
int64_t kvCacheRegion::allocCacheBlocks(uint64_t blocks, uint64_t * used_pages, hipStream_t stream) {
  uint64_t size = blocks * this->block_size;

  int64_t toallocPages = -1; 

  // Align the new offset to page_size
  uint64_t alignedSize = roundup(size, this->page_size); 

  this->total_pages = alignedSize/this->page_size;

  // Updating the offset as we are using more blocks here. 
  this->alignedSize = alignedSize;
  
  // Check how many pages should we allocated this time
  char * alignedAddr = this->dptr + alignedSize; 
  if( alignedAddr > this->nextUnmapedAddr) {

    // Check whether alignedAddr is actually aligned well
    assert((alignedAddr - this->nextUnmapedAddr)%this->page_size == 0);
    toallocPages = (alignedAddr - this->nextUnmapedAddr)/this->page_size; 

    assert(toallocPages >= 0);

    uint64_t allocSize = toallocPages * this->page_size;

    // Allocate physical pages, which will exit if can't allocate successfully
    if (toallocPages > 0 && allocatePhyPages(this->nextUnmapedAddr, allocSize) == 0) {
      //fprintf(stderr, "blocks %ld this->block_size %ld size %lx allocSize %lx toallocPages %ld this->nextUnmapedAddr %p this->page_size %ld\n", blocks, this->block_size, size, allocSize, toallocPages, this->nextUnmapedAddr, this->page_size);
      
      // Touch newly-allocates pages in order to initiate physical page allocation
      // This is important to avoid the memory allocation overhead on the critical path. 
      for(int i = 0; i < toallocPages; i++) {
        int64_t h_data = 0;
        int64_t offset = this->page_size * i;
        // Using different APIs for asynchronous memory allocations. 
        if(stream == nullptr) 
          hipMemcpyHtoD(reinterpret_cast<hipDeviceptr_t>(this->nextUnmapedAddr + offset), &h_data, sizeof(int64_t));
        else
          hipMemcpyAsync(reinterpret_cast<void *>(this->nextUnmapedAddr + offset), &h_data, sizeof(int64_t), hipMemcpyHostToDevice, stream);
      }

      this->nextUnmapedAddr = alignedAddr;
      // Update the used pages correspondingly. The statement works even when this->offset is not aligned to page_size
      *used_pages += toallocPages; 
    }
  }
 
  return toallocPages; 
}

void kvCacheRegion::freeAllPhyMemory(void) {
  freePhysicalMemory(this->dptr, this->alignedSize);
  this->offset = 0;
  this->nextUnmapedAddr = this->dptr; 
}

// freeUnusedPages from a region, and return freed pages
int kvCacheRegion::freeUnusedPages(void) {
  int freedPages = 0;

  // Free pages only when total_pages is larger than used_pages
  if(this->total_pages > this->used_pages) {
    assert(this->nextUnmapedAddr > (this->dptr + offset));

    // Get the offset of next page, since we can't collect a page if its partialy used
    uint64_t alignedSize = roundup(offset, this->page_size);
    
    // startAddr points to the beginning of the next page
    char * startAddr = this->dptr + alignedSize; 

    uint64_t size = this->nextUnmapedAddr - startAddr; 
    assert((size % this->page_size) == 0); 

    freedPages = size/this->page_size; 
    // free all unused pages of this region. 
    // If a page is partially used, then it cannot be freed 
    if(size > 0) {
      freePhysicalMemory(startAddr, size);
      this->total_pages -= freedPages;
      this->nextUnmapedAddr = startAddr;  
      // No need to change offset here. 
    } 
  }

  return freedPages; 
}

/*
** kvCacheAllocator functions implementation
*/
kvCacheAllocator::kvCacheAllocator(int64_t max_seq_length, int64_t layers_num, int64_t heads_num, int64_t head_size, int64_t tokens_per_block, int64_t dtype_size) {
  uint64_t key_cache_block_per_layer =  tokens_per_block * heads_num * head_size * dtype_size; 
  uint64_t value_cache_block_per_layer = key_cache_block_per_layer;
  uint64_t cache_block_size = (key_cache_block_per_layer + value_cache_block_per_layer) * layers_num; 

  fprintf(stderr, "kvCacheAllocator initialization: key_cache_block_per_layer-%d, cache_block_size-%lx\n", key_cache_block_per_layer, cache_block_size); 
  // Getting the cuda device and force the initialization
  hipDevice_t dev; // device
  CHECK_RT(hipFree(0));  // Force and check the initialization of the runtime
  CHECK_DRV(hipCtxGetDevice(&dev));
  
  size_t aligned_sz; 
  //_prop.type = hipMemAllocationTypeMax;
  _prop.type = hipMemAllocationTypePinned;
  //_prop.type = CU_MEM_ALLOCATION_TYPE_PORTABLE;
  _prop.location.type = hipMemLocationTypeDevice;
  _prop.location.id = dev;
  _accessDescr.flags = hipMemAccessFlagsProtReadWrite;
  _accessDescr.location = _prop.location;

  CHECK_DRV(hipMemGetAllocationGranularity(&aligned_sz, &_prop, hipMemAllocationGranularityMinimum));
  
  uint64_t max_blocks = roundup(max_seq_length, tokens_per_block)/tokens_per_block; 
  uint64_t region_size = max_blocks * cache_block_size * 2; 

  this->page_size = aligned_sz;
  this->region_size = ((region_size + aligned_sz - 1) / aligned_sz) * aligned_sz;
  this->block_size = cache_block_size;

  //printf("kvCacheAllocator: page_size-%ld, region_size-%ld, block_size-%ld\n", this->page_size, this->region_size, this->block_size);

  // TODO: finding out how much physical blocks it includes. This is just for the reference or watermark, as 
  // there is no need to rely on pre-assigned values if physical blocks are allocated on-demand
  size_t freeMem, totalMem;
  CHECK_RT(hipMemGetInfo(&freeMem, &totalMem)); 

  this->watermark_pages = (((uint64_t)(freeMem * KV_UTILIZATION_RATE))/this->page_size);  
   
  // Doing other initialization
  this->total_pages = 0;
  this->used_pages = 0;
  this->active_regions = 0;

  this->manager_running = false;
  hipCtxGetCurrent(&origContext);

  hipStreamCreate(&stream);

  // Initialize of mutex lock and condition
  pthread_mutex_init(&mutex_manager, NULL); 
  pthread_cond_init(&cond_manager, NULL); 
  manager_running = false; 

  pthread_attr_t attr; 
  pthread_attr_init(&attr);
  // Set the thread to be detached
  pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_DETACHED);

  int result = pthread_create(&this->thread_id, &attr, kvCacheAllocator::memoryManagerThread, this);
  if(result != 0) {
    fprintf(stderr, "thread creation failed!"); 
    exit(0); 
  }
}

int64_t kvCacheAllocator::getPageSize() {
  return this->page_size;
}


// reserve function, reserve virtual address space for a request
int64_t kvCacheAllocator::reserveRegion(int64_t region_id) {
  hipDeviceptr_t ptr;
  kvCacheRegion * region = nullptr;

  // Check whether there are some cached regions 
  if(this->cached_regions.size()) {
    // Pop the latest region from cached vector, which is more efficient and therefore it is the default method
    region = _getLastCachedRegion();  
  }
  else {
    // The expensive way to get a new region. Only invoked when no cached regions
    // Allocate the virtual address for this region
    CHECK_DRV(hipMemAddressReserve(&ptr, this->region_size, 0ULL, 0ULL, 0ULL));

    // Create a new region from the scratch
    region = new kvCacheRegion(this->region_size, this->block_size, this->page_size, ptr);
  }

  // Allocate one block the first region
  if(region_id == 0) {
    uint64_t total_pages; 
    region->allocCacheBlocks(1, &total_pages, nullptr); 
  }

  std::lock_guard<std::mutex> lock(this->mutex);
  
  // Record the region information
  this->active_regions += 1; 
  this->active_regions_map[region_id] = region; 

  return static_cast<int64_t>(ptr);
}

// Release the region with the given region_id
void kvCacheAllocator::_releaseRegion(int64_t region_id) {
  // Find the region corresponding to the given region_id
  if(this->active_regions_map.count(region_id) == 0) {
    fprintf(stderr, "ERROR in release: region_id-%ld does not exist at all.!\n", region_id);
    exit(-1); 
  }

  std::lock_guard<std::mutex> lock(this->mutex);

  kvCacheRegion * region = this->active_regions_map[region_id];

  // Note that as we don't actually release physical cache blocks. 
  // Therefore, we don't need to change the active_blocks here. 
  region->freeAllPhyMemory(); 
  fprintf(stderr, "Release region %d, dptr %p, aligned_size %lx\n", region_id, region->dptr, region->alignedSize);
  
  // Cache the given region, as it can be used for the future ideally. 
  // In order to reduce the overhead of memory management, we did not 
  // reclaim physical blocks until necessary.
  //_cacheReleasedRegion(region); 
}

// Cache the released region. Don't release the virtual address and physical cache blocks
void kvCacheAllocator::_cacheReleasedRegion(kvCacheRegion * region) {
  this->cached_regions.push_back(region);
}

// Get the lastly-released region. If the region has some physical blocks, 
// they will be re-utilized as well.
// Note that using cached regions is way more efficient than allocating a new region
kvCacheRegion * kvCacheAllocator::_getLastCachedRegion(void) {
  assert(!this->cached_regions.empty());

  kvCacheRegion * region = this->cached_regions.back(); 
  this->cached_regions.pop_back(); 

  return region; 
} 

// This function is invoked when the number of physical pages is above 
// the preset threshold. It performs the garbage collecton of physical pages
void kvCacheAllocator::_gcPhyPages(int64_t toCollectPages) {

  assert(toCollectPages > 0); 

  // first, collect the pages in cached regions. 
  kvCacheRegion * region; 

  // First, collect pages from cached_regions as it won't affect active requests. 
  while(!this->cached_regions.empty() && toCollectPages > 0) {
    // Release Least-Recently-Used regions at first
    region = this->cached_regions.front();
    this->cached_regions.pop_front();

    int pages = region->getAllocPhyPages();
    if(pages > 0) {
      this->total_pages -= pages; 
      toCollectPages -= pages; 
    }

    // deconstruct this region, which will collect all physical pages inside
    delete region;
  }

  // Check active regions if necessary
  while(toCollectPages > 0) {
    // Collect pages from active regions
    for(auto it = this->active_regions_map.begin(); it != this->active_regions_map.end(); it++) {
      // it->second points to the region
      region = it->second; 

      int pages = region->freeUnusedPages(); 
      if(pages > 0) {
        // Update the total_pages for the allocator
        this->total_pages -= pages; 

        toCollectPages -= pages; 
      }

      // Exit the loop if we collect enough pages
      if(toCollectPages <= 0) {
        break; 
      }
    }
  }
  
}

// alloc function, allocate physical memory, map to the reserved virtual address
// This function is designed for both prefill and decoding phase, where prefill may 
// require to save KV cache of multiple tokens, which should not invoke this function multiple times. 
// Similarly, the python code may get the physical blocks for multiple tokens during the decoding phase
// Note that the allocator doesn't care about tokens (which should be handled by the python code), but only blocks here.
int64_t kvCacheAllocator::_allocCacheBlocksForRequest(int64_t region_id, int64_t blocks, hipStream_t stream) {
  int64_t pages = -1;

  hipError_t result = hipCtxSetCurrent(origContext);
  if (result != hipSuccess) {
      std::cerr << "Failed to set CUDA context in new thread: " << result << std::endl;
      return -1;
  }

  // Find the region corresponding to the given region_id, which should reserveRegion before
  // If the region_id doesn't exist at all, it is the bug that should be fixed.  
  if(this->active_regions_map.count(region_id) == 0) {
    fprintf(stderr, "ERROR in allocation: region_id %ld does not exist at all!\n", region_id);
    exit(-1); 
  }

  std::lock_guard<std::mutex> lock(this->mutex);

  kvCacheRegion * region = this->active_regions_map[region_id]; 

  pages = region->allocCacheBlocks(blocks, &this->used_pages, stream);

  if(pages > 0) { 
    this->total_pages += pages;

    // check whether we need to purge physical memory
    if(this->total_pages >= this->watermark_pages && this->total_pages > this->used_pages) {
      int toCollectPages = std::min(this->total_pages - this->used_pages, this->total_pages - this->watermark_pages); 

      // Garbage collection for physical pages. 
      _gcPhyPages(toCollectPages);
    } 
  }

  return pages;
}

// Allocate cache blocks for a range of requests. Each request information will be an vector, with
// the request id as the first, and then number of blocks as the second. 
int64_t kvCacheAllocator::allocCacheBlocks(std::vector<std::vector<int64_t>> req_cache_blocks, hipStream_t stream) {
  int64_t pages = 0; 

  for(auto row : req_cache_blocks) {
    uint64_t region_id = row[0]; 
    uint64_t blocks = row[1]; 

    pages += _allocCacheBlocksForRequest(region_id, blocks, stream);
    //if (region_id == 11)
    fprintf(stderr, "allocate cache blocks for region-%d blocks %ld DONE\n", region_id, blocks);
  }
  //hipDeviceSynchronize(); 

  return pages; 
}


void * kvCacheAllocator::memoryManagerThread(void * arg) {
  kvCacheAllocator * instance = static_cast<kvCacheAllocator *>(arg); 

  while(true) {
    pthread_mutex_lock(&instance->mutex_manager); 

    // We will wait if manager_running is true (didn't finish last memory management operations)
    // or there is no need to perform memory management
    while(!instance->manager_running) {
      pthread_cond_wait(&instance->cond_manager, &instance->mutex_manager); 
    }
  
    // Perform memory management asynchronously
    instance->releaseRegions(instance->free_caches);
    instance->allocCacheBlocks(instance->req_cache_blocks, instance->stream);

    //pthread_mutex_lock(&instance->mutex_manager); 
    instance->manager_running = false; 
    pthread_cond_signal(&instance->cond_manager);
    pthread_mutex_unlock(&instance->mutex_manager); 
  }

  return NULL;
}
/* 
   This function mainly sets the work to be done, and then notify the manager thread to 
   perform memory management asynchronously. 
 */
void kvCacheAllocator::doAsyncKVCacheManage(std::vector<int64_t> free_caches, std::vector<std::vector<int64_t>> req_cache_blocks) {
    pthread_mutex_lock(&this->mutex_manager);
    
    // If the manager has not finished, waiting on the condition 
    while(this->manager_running) {
      fprintf(stderr, "waiting for the virtual memory management in asyn mode\n"); 
      pthread_cond_wait(&this->cond_manager, &this->mutex_manager); 
    }

    this->free_caches.clear(); 
    this->req_cache_blocks.clear(); 

    // Copying the work to the shared area
    for(auto cache_id: free_caches) {
      //fprintf(stderr, "releasing cache_id %d\n", cache_id); 
      this->free_caches.push_back(cache_id); 
    }

    for(auto cache_block: req_cache_blocks) {
      this->req_cache_blocks.push_back(cache_block); 
    }
    
    this->manager_running = true; 
    pthread_cond_signal(&this->cond_manager); 
    pthread_mutex_unlock(&this->mutex_manager);
}

void kvCacheAllocator::updateCacheBlocks(bool immediate_allocate, std::vector<int64_t> free_caches, std::vector<std::vector<int64_t>> req_cache_blocks) {
  //Py_BEGIN_ALLOW_THREADS
  //fprintf(stderr, "NNNNNNN is_prefill_phase is %d\n", is_prefill_phase); 

  if(immediate_allocate) {
    pthread_mutex_lock(&this->mutex_manager);
    
    // If the manager has not finished, waiting on the condition 
    while(this->manager_running) {
      pthread_cond_wait(&this->cond_manager, &this->mutex_manager); 
    }
    this->releaseRegions(free_caches);
    this->allocCacheBlocks(req_cache_blocks, nullptr);

    pthread_mutex_unlock(&this->mutex_manager); 
  }
  else {
    doAsyncKVCacheManage(free_caches, req_cache_blocks);
  }
  //Py_END_ALLOW_THREADS
}

// Release regions specified in the vector
void kvCacheAllocator::releaseRegions(std::vector<int64_t> regions) {
  for(auto region : regions) {
    _releaseRegion(region);
  }
}


int64_t kvCacheAllocator::getAllocPhyPages(int64_t region_id) {
  int64_t pages = 0; 

  if(region_id == 0) {
    pages = this->total_pages; 
  }
  else {
    // Find the region corresponding to the given region_id, which should reserveRegion before
    // If the region_id doesn't exist at all, it is the bug that should be fixed.  
    if(this->active_regions_map.count(region_id) == 0) {
      fprintf(stderr, "ERROR: region_id does not exist at getAllocPhyPages.!");
      exit(-1); 
    }

    std::lock_guard<std::mutex> lock(this->mutex);

    kvCacheRegion * region = this->active_regions_map[region_id]; 
    pages = region->getAllocPhyPages(); 
  }

  return pages;
}

void kvCacheAllocator::collectPhyPages(int64_t pages) {
  if(pages == 0) {
    // Collect pages defined by watermark
    pages = std::min(this->total_pages - this->used_pages, this->total_pages - this->watermark_pages); 
  }
  
  _gcPhyPages(pages);
  return; 
}


// Swap out the caches listed in src_to_dests (from Device to Host)
void kvCacheAllocator::swapOutCache(std::vector<std::vector<int64_t>> src_to_dests) {
  
  for(auto item: src_to_dests) {
    int64_t region_id = item[0]; 
    int64_t dest_ptr = item[1]; 
    int64_t size = item[2]; 

    kvCacheRegion * region = this->active_regions_map[region_id];
    void * src_ptr = region->getStartPtr(); 

    hipMemcpy(reinterpret_cast<void*>(dest_ptr), reinterpret_cast<const void*>(src_ptr),
                    size, hipMemcpyDeviceToHost);

    // After reading, now releasing the region's memory in order to free memory for other requests
    region->freeAllPhyMemory(); 
    fprintf(stderr, "Swapped out region %d, dptr %p, aligned_size %lx\n", region_id, region->dptr, region->alignedSize);
 
  }
}

// Swap in the caches listed in src_to_dests (from Host to Device)
void kvCacheAllocator::swapInCache(std::vector<std::vector<int64_t>> src_to_dests) {
    
  for(auto item: src_to_dests) {
    int64_t src_ptr = item[0]; 
    int64_t region_id = item[1]; 
    int64_t blocks = item[2]; 

    // Allocate physical memory at first
    kvCacheRegion * region = this->active_regions_map[region_id];
    region->allocCacheBlocks(blocks, &this->used_pages, nullptr);

    int64_t size = blocks * this->block_size;
    void * dest_ptr = region->getStartPtr(); 
    printf("SWPAIN src_ptr %lx, regionid-%ld, blocks %ld, address: %p, size: %lx\n", src_ptr, region_id, blocks, dest_ptr, size);

    hipMemcpy(reinterpret_cast<void*>(dest_ptr), reinterpret_cast<const void*>(src_ptr),
                    size, hipMemcpyHostToDevice);
  }

}

