#include "hip/hip_runtime.h"
/*
 Copyright (c) ByteDance Inc.
 Authors: 
  - Tongping Liu (tongping.liu@bytedance.com)
 */ 
 
#include <c10/core/ScalarType.h>
#include <cstdint>
#include <cstdio>
#include <string>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>
#include <Python.h>
#include <pthread.h>

#include "dattn.h"


#define KV_UTILIZATION_RATE (0.9)

constexpr int64_t MEGABYTES=1048576;
constexpr int64_t PAGE_SIZE=(MEGABYTES * 2); 
constexpr int64_t GIGABYTES=(MEGABYTES * 1024);

/* 
  In this allocator, we only have the following concepts, but without the concept of tokens.
  The python portion should convert the number of tokens to tokens depending on their cache_block_size (e.g., 16)
  Region: virtual address space for a request. Currently, we support the space for max_seq_len.
 */
static uint64_t roundup(uint64_t size, uint64_t align_size) {
  return ((size + align_size - 1)/align_size) * align_size; 
}

using PhysicalBlock = struct {
    hipMemGenericAllocationHandle_t handle;
};

class PhysicalBlocksManager {
public:
  // Available blocks will be placed in this pool 
  std::vector<PhysicalBlock> block_pool; 
  // All in-use blocks will be placed in the map. 
  std::unordered_map<void *, PhysicalBlock> block_map;
  int64_t block_size;
  int64_t free_blocks; 
  int64_t total_size;
  int64_t max_allowed_size; // maximum allowed size for KV cache
  int64_t page_size;  
  int64_t incremental_size; 
  int64_t tofree_blocks_watermark; 
  int64_t num_tofree_blocks; 
  hipMemAllocationProp prop;

  PhysicalBlocksManager(); 
  ~PhysicalBlocksManager();
  void initialize(size_t max_allowed_size, size_t total_memory, size_t block_size);

  PhysicalBlock allocate();
  void record(void * virtual_address, PhysicalBlock block);

  void free(void * virtual_address);

  void cleanup();

private:
  void _free_blocks_from_pool(int64_t num_blocks);
  void _increase_blocks(int64_t num_blocks);

}; 



static PhysicalBlocksManager _block_manager;

/*
 * In this file, there are three concepts:
    page_size: the actual page size of the underlying hardware, typically 2MB for cuda GPU
    cache_block_size: the original size of each block (16 tokens). However, this size (e.g., 5M) may not be aligned well with pages. 
    physical_block_size: the actual size used in managing of pysical blocks, which is consisted of multiple of cache_block_size. 
                      In particular, this size is algined with page_size.  
 */
PhysicalBlocksManager::PhysicalBlocksManager() {
  this->prop = {};
  this->prop.type = hipMemAllocationTypePinned;
  this->prop.location.type = hipMemLocationTypeDevice;
  this->prop.location.id = 0;

  // Each time, the size of physical blocks is to be increased whenever no objects is available
  this->incremental_size = 2 * GIGABYTES;
  this->total_size = 0;
  this->block_size = 0; 
  this->free_blocks = 0;  
}

PhysicalBlocksManager::~PhysicalBlocksManager() {
}

void PhysicalBlocksManager::_increase_blocks(int64_t num_blocks) {
    hipError_t result;
    //fprintf(stderr, "_increase_blocks num_blocks-%d\n", num_blocks); 
    for (size_t i = 0; i < num_blocks; i++) {
        hipMemGenericAllocationHandle_t handle;
        result = hipMemCreate(&handle, this->block_size, &this->prop, 0);
        if (result != hipSuccess) {
          fprintf(stderr, "Failed to create memory allocation, i-%d, with result %ld\n", i, result);
          exit(-1);        
        }

        block_pool.emplace_back(PhysicalBlock{handle});
    }
    // Update the number of blocks
    this->free_blocks += num_blocks; 
}

void PhysicalBlocksManager::initialize(size_t max_allowed_size, size_t total_memory, size_t block_size) {
    hipError_t result;
    size_t page_size; 

    // Getting the granularity of page isze. 
    result = hipMemGetAllocationGranularity(&page_size, &this->prop, hipMemAllocationGranularityMinimum);
    if (result != hipSuccess) {
        throw std::runtime_error("Failed to get page size");
    }

    assert(page_size == PAGE_SIZE); 
    assert(total_memory % GIGABYTES == 0);

    // We assume that cache_block_size is multiple of megabytes here.  

    // Allocate the initial blocks based on user's specification
    this->block_size = block_size; 

    this->max_allowed_size = max_allowed_size; 
    this->tofree_blocks_watermark = (this->incremental_size * 2)/block_size; 
    this->num_tofree_blocks = this->incremental_size/block_size;  

    // Allocate the physical memory with specified size 
    int64_t to_allocate_memory = min(total_memory, max_allowed_size); 
    this->total_size = to_allocate_memory; 
    size_t num_blocks = to_allocate_memory / block_size;
    fprintf(stderr, "total_memory %lx, max_allowed_size %lx num_blocks-%ld\n", total_memory, max_allowed_size, num_blocks);
    _increase_blocks(num_blocks);
}

PhysicalBlock PhysicalBlocksManager::allocate(void) {
    if(this->free_blocks == 0) {
      assert(this->block_pool.size() == 0); 

      // Keeping increase the memory if the GPU memory is sufficient
      int64_t allow_size = this->max_allowed_size - this->total_size;
      int64_t alloc_size; 
      //fprintf(stderr, "alloc_size %lx allow_size %lx total_size %lx\n", alloc_size, allow_size, this->total_size);

      if(allow_size <= 0) {
        fprintf(stderr, "There is no sufficent GPU memory now. ");
        exit(0);         
      }

      if (allow_size > this->incremental_size) {
        alloc_size = this->incremental_size; 
      }
      else {
        // Less than the incremental_size. 
        alloc_size = roundup(allow_size, this->block_size);  
      }

      int64_t blocks = alloc_size/this->block_size; 

      _increase_blocks(blocks);
      this->total_size += alloc_size; 
    }

    assert(this->free_blocks > 0); 
    assert(this->block_pool.size() > 0); 

    PhysicalBlock block = block_pool.back(); 
    block_pool.pop_back();     
    this->free_blocks--; 
    return block; 
}

void PhysicalBlocksManager::free(void * virtual_address) {
  PhysicalBlock block;
  bool is_exist = false; 

  if(block_map.count(virtual_address)) {
    block = block_map[virtual_address];

    block_map.erase(virtual_address); 
    is_exist = true; 
  }
  
  if (!is_exist) {
    fprintf(stderr, "Wrong: virtual_address-%p does not exist\n", virtual_address);
    exit(-1); 
  }

  // Adding this block to the block_pool
  block_pool.push_back(block); 
  this->free_blocks += 1; 

  if(block_pool.size() > this->tofree_blocks_watermark) {
    _free_blocks_from_pool(this->num_tofree_blocks);
  }
}

void PhysicalBlocksManager::_free_blocks_from_pool(int64_t num_blocks) {
    for(int i = 0; i < num_blocks; i++) {
      PhysicalBlock block = block_pool.back(); 
      hipError_t status = hipSuccess;
      if((status = hipMemRelease(block.handle)) != hipSuccess) {
        fprintf(stderr, "hipMemRelease failed, err code: %d\n", status);
      } 
      block_pool.pop_back(); 
    }

    this->free_blocks -= num_blocks; 
}

void PhysicalBlocksManager::cleanup() {
    for (auto& block : block_pool) {
      hipError_t status = hipSuccess;
      if((status = hipMemRelease(block.handle)) != hipSuccess) {
        fprintf(stderr, "hipMemRelease failed, err code: %d\n", status);
      } 
    }
    block_pool.clear();
}

void PhysicalBlocksManager::record(void * virtual_address, PhysicalBlock block) {
  block_map[virtual_address] = block; 
}

static hipMemAccessDesc _accessDescr = {};
 

/*
** kvCacheRegion functions implementation
*/
kvCacheRegion::kvCacheRegion(int64_t cache_block_size, int64_t physical_block_size, hipDeviceptr_t ptr) {
  this->cache_block_size = cache_block_size;
  this->physical_block_size = physical_block_size;
  this->dptr = reinterpret_cast<char*>(ptr);  
  this->nextUnmappedAddr = reinterpret_cast<char*>(ptr); 
  this->mapped_size = 0;
}

// Decontructor: release all physical pages of this region
kvCacheRegion::~kvCacheRegion() {
  freeAllPhyMemory(); 
  // Note that since the region is detroyed, 
  // no need to clear other counters. 
}

hipDeviceptr_t kvCacheRegion::getStartPtr(void) {
  return reinterpret_cast<hipDeviceptr_t>(this->dptr); 
} 

/*
  kvCacheRegion function: allocate cached blocks  
    if the return value > 0, then it is succesful. 
 */ 
void kvCacheRegion::updateBlocks(uint64_t blocks, hipStream_t stream) {
  uint64_t newSize = blocks * this->cache_block_size;
  newSize = roundup(newSize, this->physical_block_size); 

  int64_t distance; 

  // No need to allocate if size is not changed
  if(newSize == this->mapped_size) {
    return; 
  }
  else if (newSize < this->mapped_size) {
    // Shrink the memory for this region
    distance = this->mapped_size - newSize; 
    int64_t blocks_num = distance/this->physical_block_size; 

    char * addr = this->dptr + newSize; 
    this->nextUnmappedAddr = addr; 

    // Unmap unnecessary memory
    hipError_t res; 
    res = hipMemUnmap(reinterpret_cast<hipDeviceptr_t>(addr), distance); 
    if(res != hipSuccess) {
      const char* errorStr;
      hipDrvGetErrorString(res, &errorStr);
      fprintf(stderr, "hipMemUnmap failed when deallocating ptr %p and size %lx with error %s\n", addr, distance, errorStr);
      exit(-1);
    }       

    //fprintf(stderr, "reduceBlocks, newSize: %lx, addr: %p, distance-%lx, blocks %ld, this->mapped_size: %lx \n", newSize, addr, distance, blocks_num, this->mapped_size);
    for(int i = 0; i < blocks_num; i++) {
      // Free the actual physical memory by putting it back to the pool
      _block_manager.free(addr); 

      addr += this->physical_block_size; 
    }
  }
  else {
    // Increase the memory for this region
    distance = newSize - this->mapped_size; 
    int64_t blocks_num = distance/this->physical_block_size; 

    char * addr = this->nextUnmappedAddr;

    //hipDeviceSynchronize();

    // Map new memory
    hipError_t res; 
    int64_t size = this->physical_block_size;  
    //fprintf(stderr, "increaseBlocks newSize: %lx, addr: %p, distance-%lx, blocks %ld, this->mapped_size: %lx\n", newSize, addr, distance, blocks_num, this->mapped_size);
    for(int i = 0; i < blocks_num; i++) {
      // Allocate a physical block 
      PhysicalBlock block = _block_manager.allocate();
      if ((res = hipMemMap(reinterpret_cast<hipDeviceptr_t>(addr), size, 0ULL, block.handle, 0ULL)) == hipSuccess) {
        if ((res = hipMemSetAccess(reinterpret_cast<hipDeviceptr_t>(addr), size, &_accessDescr, 1)) != hipSuccess) {
          fprintf(stderr, "hipMemMap success,but hipMemSetAccess failed!, err code: %d\n", res);
          hipMemUnmap(reinterpret_cast<hipDeviceptr_t>(addr), size);
          exit(-1);
        }
      }
      else {
        const char* errorStr;
        hipDrvGetErrorString(res, &errorStr);
        fprintf(stderr, "hipMemMap failed when deallocating ptr %p res %d with error %s\n", addr, res, errorStr);
      }

      _block_manager.record(addr, block); 

      // Update addr to the next block
      addr += this->physical_block_size; 
    }
    this->nextUnmappedAddr = addr; 
  }

  this->mapped_size = newSize; 
}

void kvCacheRegion::freeAllPhyMemory(void) {
  //fprintf(stderr, "freeAllPhyMemory dtpr %p mapped_size %lx\n", this->dptr, this->mapped_size);
  assert (this->mapped_size > 0);

  char * addr = this->dptr;
  hipError_t res = hipMemUnmap(reinterpret_cast<hipDeviceptr_t>(addr), this->mapped_size); 
  if(res != hipSuccess) {
    const char* errorStr;
    hipDrvGetErrorString(res, &errorStr);
    fprintf(stderr, "hipMemUnmap failed when deallocating ptr %p and size %lx with error %s\n", reinterpret_cast<hipDeviceptr_t>(addr), this->mapped_size, errorStr);
    exit(-1);
  }       

  int64_t blocks_num = this->mapped_size/this->physical_block_size; 
  for(int i = 0; i < blocks_num; i++) {
    // Free the actual physical memory by putting it back to the pool
    _block_manager.free(addr); 

    addr += this->physical_block_size; 
  }

  // Note that we don't actually release virtual memory (hipMemAddressFree)
  this->nextUnmappedAddr = this->dptr; 
  this->mapped_size = 0; 
}

/*
** kvCacheAllocator functions implementation
* TODO: we may need to remove some details from the allocator, such as max_seq_length, layers_num. 
*       But instead, we should add the initial allocation size, or we can use number of blocks (allocated size, so that )
*/
kvCacheAllocator::kvCacheAllocator(int64_t max_gpu_memory_size, int64_t cache_block_size, int64_t region_cache_size) {
  hipDevice_t device;
  //CHECK_DRV(hipInit(0));
  CHECK_RT(hipFree(0));
  // Getting the cuda device and force the initialization
  CHECK_DRV(hipCtxGetDevice(&device));

  CHECK_DRV(hipCtxGetCurrent(&this->torchContext)); 
  //CHECK_DRV(hipCtxCreate(&this->origContext, 0, device));
  //CHECK_DRV(hipCtxSetCurrent(this->origContext));

  size_t free_memory, total_memory;
  CHECK_RT(hipMemGetInfo(&free_memory, &total_memory)); 
  fprintf(stderr, "free_memory-%lx, total_memory-%lx\n", free_memory, total_memory);

  hipError_t result;
  size_t page_size; 

  hipMemAllocationProp prop = {};
  prop.type = hipMemAllocationTypePinned;
  prop.location.type = hipMemLocationTypeDevice;
  prop.location.id = 0;

  _accessDescr.flags = hipMemAccessFlagsProtReadWrite;
  _accessDescr.location = prop.location;

  // Getting the granularity of page isze. 
  result = hipMemGetAllocationGranularity(&page_size, &prop, hipMemAllocationGranularityMinimum);
  if (result != hipSuccess) {
    fprintf(stderr, "Failed to get page size!\n");
    exit(-1);
  }

  fprintf(stderr, "kvCacheAllocator: max_gpu_memory_size - %lx, cache_block_size - %lx, region_cache_size - %lx, page_size - %lx\n", max_gpu_memory_size, cache_block_size, region_cache_size, page_size);
  assert(page_size == PAGE_SIZE); 

  int64_t to_allocate_memory = 2 * GIGABYTES; 
  if(free_memory < to_allocate_memory) {
    fprintf(stderr, "Insufficient gpu memory\n");
    exit(-1);
  }

  int64_t physical_block_size = cache_block_size; 

  while(physical_block_size%page_size != 0) {
    physical_block_size *= 2; 

    // Adding an explicit checking. 
    if(physical_block_size > 40*MEGABYTES) {
      fprintf(stderr, "Invalid physical_block_size %lx, with cache_block_size-%lx!!", physical_block_size, cache_block_size);
      exit(-1);
    }
  }
  this->physical_block_size = physical_block_size; 

  // Initialize block manager
  // max_allowed_size should be related to num_blocks, initialized GPU memory, cache_block_size
  _block_manager.initialize(max_gpu_memory_size, to_allocate_memory, physical_block_size);

  this->page_size = PAGE_SIZE;
  this->region_size = region_cache_size;
  this->cache_block_size = cache_block_size;

  this->manager_running = false;

  // Initialize of mutex lock and condition
  pthread_mutex_init(&mutex_manager, NULL); 
  pthread_cond_init(&cond_manager, NULL); 
  manager_running = false; 

  pthread_attr_t attr; 
  pthread_attr_init(&attr);
  // Set the thread to be detached
  pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_DETACHED);

  if(pthread_create(&this->thread_id, &attr, kvCacheAllocator::memoryManagerThread, this) != 0) {
    fprintf(stderr, "thread creation failed!"); 
    exit(-1); 
  }

  CHECK_DRV(hipCtxSetCurrent(this->torchContext));
}

int64_t kvCacheAllocator::getPageSize() {
  return this->page_size;
}


// reserve function, reserve virtual address space for a request
int64_t kvCacheAllocator::reserveRegion(int64_t region_id) {
  hipDeviceptr_t ptr;
  kvCacheRegion * region = nullptr;

  // The expensive way to get a new region. Only invoked when no cached regions
  // Allocate the virtual address for this region
  CHECK_DRV(hipMemAddressReserve(&ptr, this->region_size, 0ULL, 0ULL, 0ULL));

  // Create a new region from the scratch
  region = new kvCacheRegion(this->cache_block_size, this->physical_block_size, ptr);

  // Allocate one block the first region
  if(region_id == 0) {
    region->updateBlocks(1, nullptr); 
  }

  // Record the region information
  this->active_regions_map[region_id] = region; 

  return static_cast<int64_t>(ptr);
}

std::vector<int64_t> kvCacheAllocator::allocCPUCaches(int64_t num_caches, int64_t cache_size) {
  std::vector<int64_t> cache_addresses; 
  
  for(int i = 0; i < num_caches; i++) {
    void * address;  
    hipError_t err = hipHostAlloc(&address, cache_size, hipHostMallocDefault);
    if (err != hipSuccess) {
        std::cerr << "hipHostAlloc failed: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
    cache_addresses.push_back((int64_t)address); 
  }

  return cache_addresses; 
}

// Release the region with the given region_id
void kvCacheAllocator::_releaseRegion(int64_t region_id) {
  // Find the region corresponding to the given region_id
  if(this->active_regions_map.count(region_id) == 0) {
    fprintf(stderr, "ERROR in release: region_id-%ld does not exist at all.!\n", region_id);
    exit(-1); 
  }

  //std::lock_guard<std::mutex> lock(this->mutex);
  kvCacheRegion * region = this->active_regions_map[region_id];

  //fprintf(stderr, "before release region %ld, blocks %d\n", region_id, _block_manager.block_pool.size()); 
  // Note that as we don't actually release physical cache blocks. 
  // Therefore, we don't need to change the active_blocks here. 
  region->freeAllPhyMemory();
  //fprintf(stderr, "after release region %ld, blocks %d\n", region_id, _block_manager.block_pool.size()); 
  //fprintf(stderr, "release region %ld\n", region_id); 
}


// Allocate cache blocks for a range of requests. Each request information will be an vector, with
// the request id as the first, and then number of blocks as the second. 
void kvCacheAllocator::updateBlocks(std::vector<std::vector<int64_t>> update_blocks, hipStream_t stream) {
  for(auto row : update_blocks) {
    uint64_t region_id = row[0]; 
    uint64_t blocks = row[1]; 

    //fprintf(stderr, "region-%ld allocates %ld blocks. free_blocks-%d\n", region_id, blocks, _block_manager.block_pool.size()); 
    assert(this->active_regions_map.count(region_id) > 0);
    kvCacheRegion * region = this->active_regions_map[region_id];
    region->updateBlocks(blocks, stream);
    //fprintf(stderr, "after region-%ld allocates %ld blocks. free_blocks-%ld\n", region_id, blocks, _block_manager.block_pool.size()); 
    //fprintf(stderr, "region-%ld allocates %ld blocks. physical block size:%lx\n", region_id, blocks, this->physical_block_size); 
  }

  // Make sure that the asynchronized memcopy has finished. 
  if(stream)
    hipStreamSynchronize(stream);

  //fprintf(stderr, "NNNNNN after updateBlocks, handling %ld request\n", update_blocks.size());
  return; 
}

// This is a separate thread that performing both synchronous and asynchronous 
// memory management operations. 
void * kvCacheAllocator::memoryManagerThread(void * arg) {
  kvCacheAllocator * instance = static_cast<kvCacheAllocator *>(arg); 
  
  // It is required to set current context if we are going 
  //hipError_t result = hipCtxSetCurrent(instance->origContext);
  
  //hipStream_t stream = at::cuda::getCurrentCUDAStream();
  hipStream_t asyncStream;
  hipStreamCreate(&asyncStream);

  while(true) {
    pthread_mutex_lock(&instance->mutex_manager); 

    // We will wait if manager_running is true (didn't finish last memory management operations)
    // or there is no need to perform memory management
    while(!instance->manager_running) {
      pthread_cond_wait(&instance->cond_manager, &instance->mutex_manager); 
    }

    hipStream_t stream = nullptr; 
    // We will use a different stream for asynchronous operations. 
    if(!instance->immediate_allocate) {
      //stream = asyncStream;
      stream = at::cuda::getCurrentCUDAStream();
    } 

    //fprintf(stderr, "NNNNNNNNN in handling the request!!!!!\n");
    // Perform memory management asynchronously
    instance->swapOutCache(instance->swap_out_caches, stream);
    instance->updateBlocks(instance->update_blocks, stream);
    // Swap in cache must be done after allocating cache blocks, as 
    // we may reuse an existing cache but with the expansion of its blocks 
    instance->swapInCache(instance->swap_in_caches, stream);

    //pthread_mutex_lock(&instance->mutex_manager); 
    instance->manager_running = false; 
    pthread_cond_signal(&instance->cond_manager);
    pthread_mutex_unlock(&instance->mutex_manager); 
  }

  return NULL;
}

void kvCacheAllocator::updateCacheBlocks(bool immediate_allocate, 
                                         std::vector<std::vector<int64_t>> to_update_blocks,
                                         std::vector<std::vector<int64_t>> to_swap_out,
                                         std::vector<std::vector<int64_t>> to_swap_in) {
    //fprintf(stderr, "NNNNNNNNN in handling the request updateCacheBlocks!!!!!, immediate_allocate-%d\n", immediate_allocate);
    pthread_mutex_lock(&this->mutex_manager);
    
    // If the manager has not finished, waiting on the condition 
    while(this->manager_running) {
      //fprintf(stderr, "waiting for the virtual memory management in asyn mode\n"); 
      pthread_cond_wait(&this->cond_manager, &this->mutex_manager); 
    }

    this->update_blocks.clear();
    this->swap_out_caches.clear();  
    this->swap_in_caches.clear();  

    for(auto cache_block: to_update_blocks) {
      this->update_blocks.push_back(cache_block); 
    }

    for(auto cacheInfo: to_swap_out) {
      this->swap_out_caches.push_back(cacheInfo); 
    }

    for(auto cacheInfo: to_swap_in) {
      this->swap_in_caches.push_back(cacheInfo); 
    }    
    
    this->manager_running = true; 
    this->immediate_allocate = immediate_allocate; 
    pthread_cond_signal(&this->cond_manager); 
    pthread_mutex_unlock(&this->mutex_manager);

    if(immediate_allocate) {
      // We will wait until the manager thread finishes its job
      pthread_mutex_lock(&this->mutex_manager);
      while(this->manager_running) {
        //fprintf(stderr, "waiting for the virtual memory management in asyn mode\n"); 
        pthread_cond_wait(&this->cond_manager, &this->mutex_manager); 
      } 
      pthread_mutex_unlock(&this->mutex_manager); 
    }
}

// Release regions specified in the vector
void kvCacheAllocator::releaseRegions(std::vector<int64_t> regions) {
  for(auto region : regions) {
    //fprintf(stderr, "release region-%d\n", region); 
    _releaseRegion(region);
  }
}

// Swap out the caches listed in src_to_dests (from Device to Host)
void kvCacheAllocator::swapOutCache(std::vector<std::vector<int64_t>> swap_caches, hipStream_t stream) {
  
  for(auto item: swap_caches) {
    int64_t region_id = item[0]; 
    int64_t dest_ptr = item[1]; 
    int64_t size = item[2]; 

    assert(this->active_regions_map.count(region_id) != 0);

    kvCacheRegion * region = this->active_regions_map[region_id];
    hipDeviceptr_t src_ptr = region->getStartPtr(); 
    
    //hipMemcpyDtoH(reinterpret_cast<void*>(dest_ptr), src_ptr, size); 
    //continue;

    if(stream == nullptr) {
      hipMemcpyDtoH(reinterpret_cast<void*>(dest_ptr), src_ptr, size); 
    } else {
      //hipMemcpyAsync(reinterpret_cast<void*>(dest_ptr), reinterpret_cast<void*>(src_ptr), size, hipMemcpyDeviceToHost, nullptr);
      //hipMemcpyAsync(reinterpret_cast<void*>(dest_ptr), reinterpret_cast<void*>(src_ptr), size, hipMemcpyDeviceToHost, stream);
      hipMemcpyDtoHAsync(reinterpret_cast<void*>(dest_ptr), src_ptr, size, stream); 

      //fprintf(stderr, "swaping out region-%ld, src-%p, dest-%p, size-%lx\n", region_id, src_ptr, dest_ptr, size);
    }
    
  }
}

// Swap in the caches listed in swap_caches (from Host to Device)
void kvCacheAllocator::swapInCache(std::vector<std::vector<int64_t>> swap_caches, hipStream_t stream) {
    
  for(auto item: swap_caches) {
    int64_t src_ptr = item[0]; 
    int64_t region_id = item[1]; 
    int64_t blocks = item[2]; 

    // Allocate physical memory at first
    kvCacheRegion * region = this->active_regions_map[region_id];

    int64_t size = blocks  * this->cache_block_size; 

    //fprintf(stderr, "SWPAIN allocation regionid-%ld, blocks %ld, size: %lx\n", region_id, blocks, size);
    // NOTE: no need to updateBlocks, as we have done that before
    //region->updateBlocks(blocks, stream);
    
    hipDeviceptr_t dest_ptr = region->getStartPtr(); 

    if(stream == nullptr) {
      hipMemcpyHtoD(dest_ptr, reinterpret_cast<const void*>(src_ptr), size);
    }
    else {
      hipMemcpyHtoDAsync(dest_ptr, reinterpret_cast<const void*>(src_ptr), size, stream);
    }
  }
}