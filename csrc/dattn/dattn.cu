/*
 Copyright (c) ByteDance Inc.
 Authors: 
  - Tongping Liu (tongping.liu@bytedance.com)
 */ 
 
#include <c10/core/ScalarType.h>
#include <cstdint>
#include <cstdio>
#include <string>
#include <hip/hip_runtime.h>
#include <Python.h>
#include <pthread.h>

#include "dattn.h"


#define KV_UTILIZATION_RATE (0.9)

static hipMemAllocationProp _prop = {};
static hipMemAccessDesc _accessDescr = {};
 
/* 
  In this allocator, we only have the following concepts, but without the concept of tokens.
  The python portion should convert the number of tokens to tokens depending on their block_size (e.g., 16)
  Region: virtual address space for a request. Currently, we support the space for max_seq_len.
 */
static uint64_t roundup(uint64_t size, uint64_t align_size) {
  return ((size + align_size - 1)/align_size) * align_size; 
}

static int allocatePhyPages(void * ptr, uint64_t size) {
  hipDeviceptr_t dptr = (hipDeviceptr_t)ptr;

  hipDevice_t dev; // device
  CHECK_DRV(hipCtxGetDevice(&dev));
  _prop.location.id = dev;
  _accessDescr.location = _prop.location;

  hipError_t status = hipSuccess;
  hipMemGenericAllocationHandle_t allocationHandle;
  if ((status = hipMemCreate(&allocationHandle, size, &_prop, 0)) == hipSuccess) {
    if ((status = hipMemMap(dptr, size, 0ULL, allocationHandle, 0ULL)) == hipSuccess) {
      if ((status = hipMemSetAccess(dptr, size, &_accessDescr, 1)) != hipSuccess) {
        fprintf(stderr, "hipMemMap success,but hipMemSetAccess failed!, err code: %d\n", status);
        hipMemUnmap(dptr, size);
      }
    }
    // always release the handle, but the memory is accessible util hipMemUnmap
    if((status = hipMemRelease(allocationHandle)) != hipSuccess) {
      fprintf(stderr, "hipMemRelease failed, err code: %d\n", status);
    } 
  } else {
    fprintf(stderr, "hipMemCreate %lx failed!, err code: %d\n", size, status);
  }
  return status == hipSuccess ? 0 : -1;
}

// Free the physical memory [ptr, ptr + size]
static void freePhysicalMemory(void* ptr, size_t size) {
  hipDeviceptr_t dptr = (hipDeviceptr_t)ptr;
  hipError_t res = hipMemUnmap(dptr, size); 
  if(res != hipSuccess) {
    const char* errorStr;
    hipDrvGetErrorString(res, &errorStr);
    fprintf(stderr, "hipMemUnmap failed when deallocating ptr %p and size %lx with error %s\n", ptr, size, errorStr);
  } 
}

/*
** kvCacheRegion functions implementation
*/
kvCacheRegion::kvCacheRegion(uint64_t region_size, uint64_t block_size, uint64_t page_size, hipDeviceptr_t ptr) {
  this->region_size = region_size;
  this->block_size = block_size;
  this->page_size = page_size; 
  this->dptr = reinterpret_cast<char*>(ptr);  
  this->nextUnmapedAddr = reinterpret_cast<char*>(ptr); 

  this->total_pages = 0;
  this->used_pages = 0; 
  this->alignedSize = 0;
}

// Decontructor: release all physical pages of this region
kvCacheRegion::~kvCacheRegion() {
  freeAllPhyMemory(); 
  // Note that since the region is detroyed, 
  // no need to clear other counters. 
}

hipDeviceptr_t kvCacheRegion::getStartPtr(void) {
  return reinterpret_cast<hipDeviceptr_t>(this->dptr); 
} 

/*
  kvCacheRegion function: allocate cached blocks  
    if the return value > 0, then it is succesful. 
 */ 
int64_t kvCacheRegion::allocCacheBlocks(uint64_t blocks, uint64_t * used_pages, hipStream_t stream) {
  uint64_t size = blocks * this->block_size;

  int64_t toallocPages = -1; 

  // Align the new offset to page_size
  uint64_t alignedSize = roundup(size, this->page_size); 

  // No need to allocate if the cache is already larger than what we need
  if(alignedSize < this->alignedSize) {
    return 0; 
  }

  this->total_pages = alignedSize/this->page_size;

  // Updating the offset as we are using more blocks here. 
  this->alignedSize = alignedSize;
  
  // Check how many pages should we allocated this time
  char * alignedAddr = this->dptr + alignedSize; 
  if( alignedAddr > this->nextUnmapedAddr) {

    // Check whether alignedAddr is actually aligned well
    assert((alignedAddr - this->nextUnmapedAddr)%this->page_size == 0);
    toallocPages = (alignedAddr - this->nextUnmapedAddr)/this->page_size; 

    assert(toallocPages >= 0);

    uint64_t allocSize = toallocPages * this->page_size;

    // Allocate physical pages, which will exit if can't allocate successfully
    if (toallocPages > 0 && allocatePhyPages(this->nextUnmapedAddr, allocSize) == 0) {
      
      // Touch newly-allocates pages in order to initiate physical page allocation
      // This is important to avoid the memory allocation overhead on the critical path. 
      for(int i = 0; i < toallocPages; i++) {
        int64_t h_data = 0;
        int64_t offset = this->page_size * i;
        // Using different APIs for asynchronous memory allocations. 
        if(stream == nullptr) { 
          hipMemcpyHtoD(reinterpret_cast<hipDeviceptr_t>(this->nextUnmapedAddr + offset), &h_data, sizeof(int64_t));
        } else {
          hipMemcpyHtoDAsync(reinterpret_cast<hipDeviceptr_t>(this->nextUnmapedAddr + offset), &h_data, sizeof(int64_t), stream);
        }
      }

      this->nextUnmapedAddr = alignedAddr;
      //fprintf(stderr, "%p : blocks %ld blocksize %lx this->alignedSize %lx this->nextUnmapedAddr %p alignedAddr %p\n", this->dptr, blocks, this->block_size, this->alignedSize, this->nextUnmapedAddr, alignedAddr);
      // Update the used pages correspondingly. The statement works even when this->offset is not aligned to page_size
      *used_pages += toallocPages; 
    }
  }
 
  return toallocPages; 
}

void kvCacheRegion::freeAllPhyMemory(void) {
  //fprintf(stderr, "freeAllPhyMemory dtpr %p alignedSize %lx\n", this->dptr, this->alignedSize);
  assert (this->alignedSize > 0);
  freePhysicalMemory(this->dptr, this->alignedSize);
  this->nextUnmapedAddr = this->dptr; 
  this->alignedSize = 0; 
}

/*
** kvCacheAllocator functions implementation
*/
kvCacheAllocator::kvCacheAllocator(int64_t max_seq_length, int64_t layers_num, int64_t heads_num, int64_t head_size, int64_t tokens_per_block, int64_t dtype_size) {
  uint64_t key_cache_block_per_layer =  tokens_per_block * heads_num * head_size * dtype_size; 
  uint64_t value_cache_block_per_layer = key_cache_block_per_layer;
  uint64_t cache_block_size = (key_cache_block_per_layer + value_cache_block_per_layer) * layers_num; 

  fprintf(stderr, "kvCacheAllocator initialization: key_cache_block_per_layer-%d, cache_block_size-%lx\n", key_cache_block_per_layer, cache_block_size); 
  // Getting the cuda device and force the initialization
  hipDevice_t dev; // device
  CHECK_RT(hipFree(0));  // Force and check the initialization of the runtime
  CHECK_DRV(hipCtxGetDevice(&dev));
  
  size_t aligned_sz; 
  //_prop.type = hipMemAllocationTypeMax;
  _prop.type = hipMemAllocationTypePinned;
  //_prop.type = CU_MEM_ALLOCATION_TYPE_PORTABLE;
  _prop.location.type = hipMemLocationTypeDevice;
  _prop.location.id = dev;
  _accessDescr.flags = hipMemAccessFlagsProtReadWrite;
  _accessDescr.location = _prop.location;

  CHECK_DRV(hipMemGetAllocationGranularity(&aligned_sz, &_prop, hipMemAllocationGranularityMinimum));
  
  uint64_t max_blocks = roundup(max_seq_length, tokens_per_block)/tokens_per_block; 
  uint64_t region_size = max_blocks * cache_block_size * 2; 

  this->page_size = aligned_sz;
  this->region_size = ((region_size + aligned_sz - 1) / aligned_sz) * aligned_sz;
  this->block_size = cache_block_size;

  //printf("kvCacheAllocator: page_size-%ld, region_size-%ld, block_size-%ld\n", this->page_size, this->region_size, this->block_size);

  // TODO: finding out how much physical blocks it includes. This is just for the reference or watermark, as 
  // there is no need to rely on pre-assigned values if physical blocks are allocated on-demand
  size_t freeMem, totalMem;
  CHECK_RT(hipMemGetInfo(&freeMem, &totalMem)); 

  this->watermark_pages = (((uint64_t)(freeMem * KV_UTILIZATION_RATE))/this->page_size);  
   
  // Doing other initialization
  this->total_pages = 0;
  this->used_pages = 0;
  this->active_regions = 0;

  this->manager_running = false;
  hipCtxGetCurrent(&origContext);

  hipStreamCreate(&stream);

  // Initialize of mutex lock and condition
  pthread_mutex_init(&mutex_manager, NULL); 
  pthread_cond_init(&cond_manager, NULL); 
  manager_running = false; 

  pthread_attr_t attr; 
  pthread_attr_init(&attr);
  // Set the thread to be detached
  pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_DETACHED);

  int result = pthread_create(&this->thread_id, &attr, kvCacheAllocator::memoryManagerThread, this);
  if(result != 0) {
    fprintf(stderr, "thread creation failed!"); 
    exit(0); 
  }
}

int64_t kvCacheAllocator::getPageSize() {
  return this->page_size;
}


// reserve function, reserve virtual address space for a request
int64_t kvCacheAllocator::reserveRegion(int64_t region_id) {
  hipDeviceptr_t ptr;
  kvCacheRegion * region = nullptr;

  // The expensive way to get a new region. Only invoked when no cached regions
  // Allocate the virtual address for this region
  CHECK_DRV(hipMemAddressReserve(&ptr, this->region_size, 0ULL, 0ULL, 0ULL));

  // Create a new region from the scratch
  region = new kvCacheRegion(this->region_size, this->block_size, this->page_size, ptr);

  // Allocate one block the first region
  if(region_id == 0) {
    uint64_t total_pages; 
    region->allocCacheBlocks(1, &total_pages, nullptr); 
  }

  std::lock_guard<std::mutex> lock(this->mutex);
  
  // Record the region information
  this->active_regions += 1; 
  this->active_regions_map[region_id] = region; 

  return static_cast<int64_t>(ptr);
}

// Release the region with the given region_id
void kvCacheAllocator::_releaseRegion(int64_t region_id) {
  // Find the region corresponding to the given region_id
  if(this->active_regions_map.count(region_id) == 0) {
    fprintf(stderr, "ERROR in release: region_id-%ld does not exist at all.!\n", region_id);
    exit(-1); 
  }

  std::lock_guard<std::mutex> lock(this->mutex);

  kvCacheRegion * region = this->active_regions_map[region_id];

  // Note that as we don't actually release physical cache blocks. 
  // Therefore, we don't need to change the active_blocks here. 
  region->freeAllPhyMemory();
  fprintf(stderr, "release region %ld\n", region_id); 
}

// alloc function, allocate physical memory, map to the reserved virtual address
// This function is designed for both prefill and decoding phase, where prefill may 
// require to save KV cache of multiple tokens, which should not invoke this function multiple times. 
// Similarly, the python code may get the physical blocks for multiple tokens during the decoding phase
// Note that the allocator doesn't care about tokens (which should be handled by the python code), but only blocks here.
int64_t kvCacheAllocator::_allocCacheBlocksForRequest(int64_t region_id, int64_t blocks, hipStream_t stream) {
  int64_t pages = -1;

  hipError_t result = hipCtxSetCurrent(origContext);
  if (result != hipSuccess) {
      std::cerr << "Failed to set CUDA context in new thread: " << result << std::endl;
      return -1;
  }

  // Find the region corresponding to the given region_id, which should reserveRegion before
  // If the region_id doesn't exist at all, it is the bug that should be fixed.  
  if(this->active_regions_map.count(region_id) == 0) {
    fprintf(stderr, "ERROR in allocation: region_id %ld does not exist at all!\n", region_id);
    exit(-1); 
  }

  std::lock_guard<std::mutex> lock(this->mutex);

  kvCacheRegion * region = this->active_regions_map[region_id]; 

  pages = region->allocCacheBlocks(blocks, &this->used_pages, stream);

  return pages;
}

// Allocate cache blocks for a range of requests. Each request information will be an vector, with
// the request id as the first, and then number of blocks as the second. 
int64_t kvCacheAllocator::allocCacheBlocks(std::vector<std::vector<int64_t>> req_cache_blocks, hipStream_t stream) {
  int64_t pages = 0; 

  for(auto row : req_cache_blocks) {
    uint64_t region_id = row[0]; 
    uint64_t blocks = row[1]; 

    pages += _allocCacheBlocksForRequest(region_id, blocks, stream);
    //if (region_id == 7)
    //fprintf(stderr, "allocate cache blocks for region-%d blocks %ld DONE\n", region_id, blocks);
  }
  //hipDeviceSynchronize(); 

  return pages; 
}


void * kvCacheAllocator::memoryManagerThread(void * arg) {
  kvCacheAllocator * instance = static_cast<kvCacheAllocator *>(arg); 

  while(true) {
    pthread_mutex_lock(&instance->mutex_manager); 

    // We will wait if manager_running is true (didn't finish last memory management operations)
    // or there is no need to perform memory management
    while(!instance->manager_running) {
      pthread_cond_wait(&instance->cond_manager, &instance->mutex_manager); 
    }
  
    // Perform memory management asynchronously
    instance->swapOutCache(instance->swap_out_caches, instance->stream);
    instance->swapInCache(instance->swap_in_caches, instance->stream);
    instance->releaseRegions(instance->free_caches);
    instance->allocCacheBlocks(instance->req_cache_blocks, instance->stream);

    //pthread_mutex_lock(&instance->mutex_manager); 
    instance->manager_running = false; 
    pthread_cond_signal(&instance->cond_manager);
    pthread_mutex_unlock(&instance->mutex_manager); 
  }

  return NULL;
}
/* 
   This function mainly sets the work to be done, and then notify the manager thread to 
   perform memory management asynchronously. 
 */
void kvCacheAllocator::doAsyncKVCacheManage(std::vector<int64_t> free_caches, std::vector<std::vector<int64_t>> req_cache_blocks, 
        std::vector<std::vector<int64_t>> to_swap_out, std::vector<std::vector<int64_t>> to_swap_in) {
    pthread_mutex_lock(&this->mutex_manager);
    
    // If the manager has not finished, waiting on the condition 
    while(this->manager_running) {
      //fprintf(stderr, "waiting for the virtual memory management in asyn mode\n"); 
      pthread_cond_wait(&this->cond_manager, &this->mutex_manager); 
    }

    this->free_caches.clear(); 
    this->req_cache_blocks.clear();
    this->swap_out_caches.clear();  
    this->swap_in_caches.clear();  

    // Copying the work to the shared area
    for(auto cache_id: free_caches) {
      //fprintf(stderr, "releasing cache_id %d\n", cache_id); 
      this->free_caches.push_back(cache_id); 
    }

    for(auto cache_block: req_cache_blocks) {
      this->req_cache_blocks.push_back(cache_block); 
    }

    for(auto cacheInfo: to_swap_out) {
      this->swap_out_caches.push_back(cacheInfo); 
    }

    for(auto cacheInfo: to_swap_in) {
      this->swap_in_caches.push_back(cacheInfo); 
    }    
    
    this->manager_running = true; 
    pthread_cond_signal(&this->cond_manager); 
    pthread_mutex_unlock(&this->mutex_manager);
}

void kvCacheAllocator::updateCacheBlocks(bool immediate_allocate, std::vector<int64_t> free_caches, 
                                         std::vector<std::vector<int64_t>> req_cache_blocks,
                                         std::vector<std::vector<int64_t>> to_swap_out,
                                         std::vector<std::vector<int64_t>> to_swap_in) {
  //Py_BEGIN_ALLOW_THREADS
  //fprintf(stderr, "NNNNNNN immediate_allocate is %d\n", immediate_allocate); 

  if(immediate_allocate) {
    pthread_mutex_lock(&this->mutex_manager);
    
    // If the manager has not finished, waiting on the condition 
    while(this->manager_running) {
      pthread_cond_wait(&this->cond_manager, &this->mutex_manager); 
    }
    this->swapOutCache(to_swap_out, nullptr);
    this->swapInCache(to_swap_in, nullptr);
    this->releaseRegions(free_caches);
    this->allocCacheBlocks(req_cache_blocks, nullptr);

    pthread_mutex_unlock(&this->mutex_manager); 
  }
  else {
    doAsyncKVCacheManage(free_caches, req_cache_blocks, to_swap_out, to_swap_in);
  }
  //Py_END_ALLOW_THREADS
}

// Release regions specified in the vector
void kvCacheAllocator::releaseRegions(std::vector<int64_t> regions) {
  for(auto region : regions) {
    //fprintf(stderr, "release region-%d\n", region); 
    _releaseRegion(region);
  }
}

// Swap out the caches listed in src_to_dests (from Device to Host)
void kvCacheAllocator::swapOutCache(std::vector<std::vector<int64_t>> swap_caches, hipStream_t stream) {
  
  for(auto item: swap_caches) {
    int64_t region_id = item[0]; 
    int64_t dest_ptr = item[1]; 
    int64_t size = item[2]; 

    kvCacheRegion * region = this->active_regions_map[region_id];
    hipDeviceptr_t src_ptr = region->getStartPtr(); 

    if(stream == nullptr) {
      hipMemcpyDtoH(reinterpret_cast<void*>(dest_ptr), src_ptr, size); 
    } else {
      hipMemcpyDtoHAsync(reinterpret_cast<void*>(dest_ptr), src_ptr, size, stream); 
    }
    
  }
}

// Swap in the caches listed in swap_caches (from Host to Device)
void kvCacheAllocator::swapInCache(std::vector<std::vector<int64_t>> swap_caches, hipStream_t stream) {
    
  for(auto item: swap_caches) {
    int64_t src_ptr = item[0]; 
    int64_t region_id = item[1]; 
    int64_t blocks = item[2]; 

    // Allocate physical memory at first
    kvCacheRegion * region = this->active_regions_map[region_id];

    int64_t size = blocks  * this->block_size; 

    region->allocCacheBlocks(blocks+1, &this->used_pages, stream);
    
    hipDeviceptr_t dest_ptr = region->getStartPtr(); 
    fprintf(stderr, "SWPAIN src_ptr %lx, regionid-%ld, blocks %ld, address: %p, size: %lx\n", src_ptr, region_id, blocks, dest_ptr, size);

    if(stream == nullptr) {
      hipMemcpyHtoD(dest_ptr, reinterpret_cast<const void*>(src_ptr), size);
    }
    else {
      hipMemcpyHtoDAsync(dest_ptr, reinterpret_cast<const void*>(src_ptr), size, stream);
    }
  }

}

